#include "hip/hip_runtime.h"
/* Copyright 2019 The TensorFlow Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in1 compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in1 writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/


#include "stupid.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"

namespace tensorflow {
namespace functor {

typedef Eigen::GpuDevice GPUDevice;

// Define the CUDA kernel.
template <typename T>
__global__ void StupidCudaKernel(const int size, const T* in, T* out) {
  if (in[0] < 0.5) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
      out[i] = 100 * in[i];
    }
  } else {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (true) {
      out[i] = 2 * in[i];
      i++;
    }
  }
}

// Define the GPU implementation that launches the CUDA kernel.
template <typename T>
struct StupidFunctor<GPUDevice, T> {
  void operator()(OpKernelContext* context, const GPUDevice& d, int size, const T* in, T* out) {
    // Launch the cuda kernel.
    //
    // See core/util/cuda_kernel_helper.h for example of computing
    // block count and thread_per_block count.
    int block_count = 1024;
    int thread_per_block = 20;
    std::cout << "Launch StupidCudaKernel" << std::endl;
    StupidCudaKernel<T><<<block_count, thread_per_block, 0, d.stream()>>>(size, in, out);
    std::cout << "Scheduled StupidCudaKernel" << std::endl;
  }
};

// Explicitly instantiate functors for the types of OpKernels registered.
template struct StupidFunctor<GPUDevice, float>;

} // namespace functor
} // namespace tensorflow
